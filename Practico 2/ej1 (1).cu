#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void read_file(const char*, int*);
int get_text_length(const char * fname);

#define A 15
#define B 27
#define M 256
#define A_MMI_M -17

#define N 512


__device__ int modulo(int a, int b){
	int r = a % b;
	r = (r < 0) ? r + b : r;
	return r;
}

__global__ void decrypt_kernel(int *d_message, int length)
{
}

int main(int argc, char *argv[])
{
	int *h_message;
	int *d_message;
	unsigned int size;

	const char * fname;

	if (argc < 2) printf("Debe ingresar el nombre del archivo\n");
	else
		fname = argv[1];

	int length = get_text_length(fname);

	size = length * sizeof(int);

	// reservar memoria para el mensaje
	h_message = (int *)malloc(size);

	// leo el archivo de la entrada
	read_file(fname, h_message);

	/* reservar memoria en la GPU */

	/* copiar los datos de entrada a la GPU */

	/* Configurar la grilla y lanzar el kernel */
	
	/* Copiar los datos de salida a la CPU en h_message */


	// despliego el mensaje
	for (int i = 0; i < length; i++) {
		printf("%c", (char)h_message[i]);
	}
	printf("\n");

	// libero la memoria en la GPU

	// libero la memoria en la CPU
	free(h_message);

	return 0;
}

	
int get_text_length(const char * fname)
{
	FILE *f = NULL;
	f = fopen(fname, "r"); //read and binary flags

	size_t pos = ftell(f);    
	fseek(f, 0, SEEK_END);    
	size_t length = ftell(f); 
	fseek(f, pos, SEEK_SET);  

	fclose(f);

	return length;
}

void read_file(const char * fname, int* input)
{
	// printf("leyendo archivo %s\n", fname );

	FILE *f = NULL;
	f = fopen(fname, "r"); //read and binary flags
	if (f == NULL){
		fprintf(stderr, "Error: Could not find %s file \n", fname);
		exit(1);
	}

	//fread(input, 1, N, f);
	int c; 
	while ((c = getc(f)) != EOF) {
		*(input++) = c;
	}

	fclose(f);
}
