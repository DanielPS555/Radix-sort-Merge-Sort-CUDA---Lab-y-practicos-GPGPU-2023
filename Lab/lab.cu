#include "hip/hip_runtime.h"
#include <stdio.h>

#include "include/lab.h"
#include <thrust/sort.h>

#define WARP_SIZE 32
#define FULL_MASK 0xffffffff
#define NONE_MASK 0x00000000
#define RADIX_SORT_BLOCK_SIZE 256

__device__
int  exlusiveScan(int valor){
    int offset = 1;
    int lane = threadIdx.x;

    // 1º Etapa
    while (offset < WARP_SIZE ) {
        int preValor = __shfl_up_sync(FULL_MASK, valor, offset);
        if ( ((lane + 1) % (offset*2)) == 0 )
            valor += preValor;
        offset *= 2;
    }

    if (lane == WARP_SIZE - 1){
        valor = 0;
    }

    // 2º Etapa
    offset /= 2;
    while (offset > 0 ) {
        int valorOffsetSuperior = __shfl_down_sync(FULL_MASK, valor, offset);
        int valorOffsetInferior = __shfl_up_sync(FULL_MASK, valor, offset);

        bool moduloDobleOffset = ((lane + 1) % (offset*2)) == 0;
        valor += moduloDobleOffset ? valorOffsetInferior : 0;
        valor = !moduloDobleOffset && ((lane + 1) % (offset)) == 0 ? valorOffsetSuperior : valor;

        offset /= 2;
    }
    return valor;
}

/**
 * @param src Arreglo a ordenar
 * @param positions Posiciones del valor inicial, si el arreglo es NULL se hace el radix sort comun
 */
__global__
void radix_sort_kernel(int * src) {
    __shared__ int swap[RADIX_SORT_BLOCK_SIZE];

    int lane = threadIdx.x % WARP_SIZE; //Se asume que los bloques son multiplos de 32, por lo que para obtener mi lane no presiso saber nada mas que mi modulo 32.
    int tie = threadIdx.x / WARP_SIZE;
    int idInArray = threadIdx.x + blockDim.x * blockIdx.x;
    int currentValue = src[idInArray];

    int valorThreadPrevio = __shfl_up_sync(FULL_MASK, currentValue, 1); // Obtengo el valor de mi lane previo
    if (lane == 0)
        valorThreadPrevio = currentValue - 1; //En caso que sea el primer lane, hago que mi "anterior" numero sea siempre menor, en particular uno menor

    int mask = 1;
    while (__all_sync(FULL_MASK,valorThreadPrevio <= currentValue) == 0){ //La operacion termina cuando el warp esta ordenado

        int notValidateMask = (currentValue & mask) == 0 ? 1 : 0;
        int valueInScan = exlusiveScan(notValidateMask);

        int totalFalse = notValidateMask + valueInScan; //Este valor es invalido para todos menos el ultimo lane, por eso en la siguiente linea todos se lo pide
        totalFalse = __shfl_sync(FULL_MASK, totalFalse, WARP_SIZE - 1); //Todos los lane obtiene el valor correcto del ultimo lane

        int t = lane - valueInScan + totalFalse;
        int nuevaPosicion = notValidateMask ? valueInScan : t;

        swap[tie + nuevaPosicion] = currentValue;

        __syncwarp();

        currentValue = swap[tie + lane];

        mask <<= 1; // Muevo la mascara

        valorThreadPrevio = __shfl_up_sync(FULL_MASK, currentValue, 1);

        if (lane == 0)
            valorThreadPrevio = currentValue - 1;
    }

    src[idInArray] = currentValue;
}


/**
 * Dado un array ordenado desde [posicionInicio, posicionInicio + size ], devuelve la posicion en la que deberia ser insertado objetoBuscado
 * Nota: si previoAIguales == true, este devuelve la posicion de forma que objetoBuscado sea insertado antes que los iguales,
 *  en caso contrario sera la posicion de ser insertado detras de los iguales
 *
 *  Retorna la posicion relativa dentro del arreglo. O sea seria la posicion real dentro del array, menos el inicio
 * @param posicionInicio
 * @param size
 * @param objetoBuscado
 * @param previoAIguales
 * @return
 */
__device__
int busquedaPorBiparticion(int * array, int posicionInicio, int size, int objetoBuscado, bool previoAIguales){

    if(size <= 0) return 0;

    int final = posicionInicio + size - 1;
    int inicio = posicionInicio;

    int medio = (inicio + final) /2;

    while (inicio < final){ //Hay mas de un elemento en el area de busqueda

        int pivot = array[medio];

        bool buscarAbajo = previoAIguales ? objetoBuscado <= pivot : objetoBuscado < pivot;

        inicio = buscarAbajo ? inicio : medio + 1;
        final  = buscarAbajo ? medio  : final;
        medio  = (inicio + final) /2;

    }

    int pivot = array[medio];

    if (previoAIguales){
        return (pivot < objetoBuscado ? medio + 1 : medio) - posicionInicio;
    } else {
        return (pivot > objetoBuscado ? medio : medio + 1) - posicionInicio;
    }

}


/**
 * Se encarga de convinar el array
 * @param arrayA
 * @param largoA
 * @param arrayB
 * @param largoB
 * @param arraySalida
 * @param sharedToUse
 */
__device__
void deviceOrderedJoin(int * src, int posicionLecturaA, int largoA, int posicionLecturaB, int largoB, int * arraySalida, int posicionSalida, int * sharedToUse){

    //En una primera instancia voy a destinar los primeros "largoPorSegmento" threads para ordenar los elementos de A, los demas para ordenar los de B.
    // ToDo utilizar la idea del ej 2 practico 4 para hacer que la lectura sea coalesed

    bool soyDeB;
    int idMiArray;
    int valor;
    if(threadIdx.x < largoA + largoB){
        soyDeB = threadIdx.x >= largoA;
        idMiArray = soyDeB ? threadIdx.x - largoA: threadIdx.x; //Es la posicion dentro de A o B respectivamente (sin contar el offset de posicionLecturaA o posicionLecturaB)
        valor = soyDeB ? src[posicionLecturaB + idMiArray] : src[posicionLecturaA + idMiArray];
        sharedToUse[threadIdx.x] = valor;
    }
    __syncthreads();

    int posicionEnElOtroArray;
    if(threadIdx.x < largoA + largoB) {
        // Nota: Aqui hay que tener un cuidado adicional, ¿Que pasa si en la misma posiicon i de A y B tenemos que r(a_i, B) = r(b_i, A)?
        //         La solucion que encontramos es hacer que ese j no sea igual discriminando entre el array A y el B, dando la presetncia a A, mas detalle a continuacion:
        //         Notar que si a_i < b_i, entonces r(a_i, B) < r(b_i,B) = i = r(a_i, A) < r(b_i, A) (analogo para b_i < a_i), entonces aqui no hay problema.
        //         Pero lo mismo no ocurre si a_i = b_i = h, ya que dependiendo de la politica de la busqueda, tanto el a_i como el b_i tendrian como r(a_i, B) = r(b_i, A) = posicion al inicio (o final) de la rafaga de h
        //         Es por eso que se propone que la politica de busqueda de r(a_i, B) y r(b_i, A) sea diferente
        //         De forma tal que los elementos de A se "inserten" previo a todos los valores iguales en B, mientras que para los de B buscaremos su posicion de forma tal que sea luego de los iguales en A
        //         Esto ultimo es lo que representa el ultimo parametro de el metodo busquedaPorBiparticion
        posicionEnElOtroArray = busquedaPorBiparticion(sharedToUse, soyDeB ? 0 : largoA, soyDeB ? largoA : largoB, valor, !soyDeB);
    }

    __syncthreads();
    if(threadIdx.x < largoA + largoB) {
        sharedToUse[idMiArray + posicionEnElOtroArray] = valor; //Escribo mi valor en la nueva posicion
    }

    __syncthreads();
    if(threadIdx.x < largoA + largoB) {
        arraySalida[posicionSalida + threadIdx.x] = sharedToUse[threadIdx.x]; //Escritura en memoria global coaleced si posicionSalida es multiplo de 32
    }
}


/**
 * En base a dos array ordenados CONSECUTIVOS de largo largoA y largoB respectivamente
 * escribe en "array" el nuevo array ordenado producto de ordenar los dos anteriores
 * @param array
 * @param largoA
 * @param largoB
 */
__global__
void orderedJoin(int * src, int largoPorSegmento){
    extern __shared__ int shared[]; //Size = 2*largoPorSegmento . Se almacena de forma compartida la informacion de los dos arrays a juntar
    int posInicioBloque = blockIdx.x * blockDim.x;
    deviceOrderedJoin(src, posInicioBloque, largoPorSegmento, posInicioBloque + largoPorSegmento, largoPorSegmento, src, posInicioBloque, shared);
}


#define BLOCK_ID  blockIdx.x  + blockIdx.y * gridDim.x
#define BLOCK_DIM blockDim.x  * blockDim.y

/**
 * One thread per separator.
 * Read each separator
 * Each block reads a sector
 * Returns 2 arrays with the A and B position of each separator
 */
__global__
void separators_kernel(int * in_data, int * out_separators_a, int * out_separators_b, const int sector_size, const int separators_per_sector, int t, int in_data_size) {
    extern __shared__ int separators[];
    int * separators_global_pos = separators + separators_per_sector;
    // based on the unique id we find the sector_id. The id of the AB sector
    const int sector_id = BLOCK_ID * blockDim.y + threadIdx.y;
    // separator in the sector
    const int separator_id = threadIdx.x;
    const bool is_a = separator_id < (separators_per_sector / 2);

    int segment_limit = min(sector_id * sector_size + (is_a ? (sector_size / 2) : sector_size), in_data_size);

    int pos = sector_id * sector_size + separator_id % (separators_per_sector / 2) * t + (is_a ? 0 : (sector_size / 2));
    // Last separator gets the last element
    if (pos >= segment_limit)
        pos = segment_limit - 1;

    int value = -1;
    if (separator_id < separators_per_sector)
    {
        value = in_data[pos];
        separators[separator_id] = value;
        separators_global_pos[separator_id] = pos;
    }

    __syncthreads();

    if (separator_id >= separators_per_sector)
        return;

    // Position in the other half of the separators array
    int s_offset = is_a ? (separators_per_sector / 2) : 0;
    int s_opp_position = busquedaPorBiparticion(separators, s_offset, separators_per_sector / 2, value, is_a);

    // if s_opp_position >= separators_per_sector / 2 then the end position is sector_id * sector_size + sector_size - 1
    // if s_opp_position <= 0 then the start position starts at the beginning of the opposite segment (A or B)
    int search_start, search_end;

    if (s_opp_position <= 0) {
        search_start = sector_id * sector_size + (is_a ? sector_size / 2 : 0);
    } else {
        search_start = separators_global_pos[s_opp_position + s_offset - 1];
    }

    if (s_opp_position >= (separators_per_sector / 2) - 1) {
        search_end = min(sector_id * sector_size + (is_a ? sector_size : sector_size / 2  ) - 1, in_data_size - 1);
    } else {
        search_end = separators_global_pos[s_opp_position + s_offset + 1];
    }

    // Find opposite position in in_data
    int opp_position = busquedaPorBiparticion(in_data, search_start, search_end - search_start, value, is_a) + search_start;

    const int pos_a = is_a ? pos : opp_position;
    const int pos_b = is_a ? opp_position : pos;

    // Ordered position in separators array out_separators_X
    int s_position = is_a ? (separator_id + s_opp_position) : (separator_id % (separators_per_sector / 2) + s_opp_position);

    out_separators_a[sector_id * separators_per_sector + s_position] = pos_a;
    out_separators_b[sector_id * separators_per_sector + s_position] = pos_b;
}

__global__
void merge_segments_kernel(int * separators_a, int * separators_b, int * src, int * dst, int sector_size) {
    __shared__ int shared[512];  // t * 2
    // Amount of segments per sector
    int segment_count = gridDim.x;
    int sector_id = blockIdx.y;
    int segment_id = blockIdx.x;  // 0-1, 1-2, 2-3, etc...

    int separator_position = sector_id * segment_count + segment_id;

    int start_a = separators_a[separator_position];
    int start_b = separators_b[separator_position];

    int sector_offset = sector_id * sector_size;

    // Repeat last element for last segment
    int end_a, end_b;
    if (segment_id == segment_count - 1) {
        end_a = sector_offset + sector_size / 2;
        end_b = sector_offset + sector_size;
    } else {
        end_a = separators_a[separator_position + 1];
        end_b = separators_b[separator_position + 1];
    }

    int dst_pos = start_a + start_b - sector_offset - sector_size / 2;

    deviceOrderedJoin(src, start_a, max(0, end_a - start_a), start_b, max(0, end_b - start_b), dst, dst_pos, shared);
}

#define MINIMUM_BLOCK_SIZE 32 // This is the minimum block size for all separator operations
void order_array(int * src_cpu, int length) {
    // 0 - Initialize GPU memory
    int * src_gpu = NULL;
    int * dst_gpu = NULL;
    int * separators_a_gpu = NULL;
    int * separators_b_gpu = NULL;

    size_t size = length * sizeof (int);
    size_t separators_size = length / MINIMUM_BLOCK_SIZE * sizeof (int);

    CUDA_CHK( hipMalloc ((void **)& src_gpu , size ) )
    CUDA_CHK( hipMalloc ((void **)& dst_gpu , size ) )
    CUDA_CHK( hipMalloc ((void **)& separators_a_gpu , separators_size ) )
    CUDA_CHK( hipMalloc ((void **)& separators_b_gpu , separators_size ) )

    CUDA_CHK ( hipMemset(separators_a_gpu, 0, separators_size) )
    CUDA_CHK ( hipMemset(separators_b_gpu, 0, separators_size) )

    CUDA_CHK( hipMemcpy(src_gpu, src_cpu, size, hipMemcpyHostToDevice))

    // 1 - Radix sort

    dim3 gridSizeRadixSort ( length / 32, 1);
    dim3 blockSizeRadixSort (32, 1);

    radix_sort_kernel<<<gridSizeRadixSort, blockSizeRadixSort>>>(src_gpu);
    CUDA_CHK(hipGetLastError())
    CUDA_CHK(hipDeviceSynchronize())

    // 2 - Merge sort on sectors smaller than a block

    int blockSize = 64; // 32 * 2;
    while (blockSize <= min(512, length)){
        dim3 gridSizeOrderedJoin ( length / blockSize, 1);
        dim3 blockSizeOrderedJoin (blockSize, 1);

        orderedJoin<<<gridSizeOrderedJoin, blockSizeOrderedJoin, blockSize*sizeof(int)>>>(src_gpu, blockSize/2);
        CUDA_CHK(hipGetLastError())
        CUDA_CHK(hipDeviceSynchronize())

        blockSize *=2;
    }

    blockSize /= 2;

    // 3 - Merge sort on sectors bigger than a block

    if (blockSize < length) {

        // Each A and B starts with the size of the block
        int sector_size = blockSize;  // Should start at 1024
        int t = blockSize / 2;

        int swapped = false;

        while (sector_size < length) {

            // 3.1 - Find separators

            sector_size *= 2;

            int sector_qty = (length + sector_size - 1) / sector_size;
            int separators_per_sector = 2*(1 + ((sector_size / 2) + t - 1)/t);

            int separators_count = sector_qty * separators_per_sector;

            int sectors_per_block = (separators_per_sector + 31) / 32;
            dim3 blockSizeFindSeparators (separators_per_sector, sectors_per_block);
            dim3 gridSizeFindSeparators ( (sector_qty + sectors_per_block - 1) / sectors_per_block, 1);

            size_t shared_size = separators_per_sector * sector_qty * sizeof(int) * 2;

            separators_kernel<<<gridSizeFindSeparators, blockSizeFindSeparators, shared_size>>>(src_gpu, separators_a_gpu, separators_b_gpu, sector_size, separators_per_sector, t, length);
            CUDA_CHK(hipGetLastError())
            CUDA_CHK(hipDeviceSynchronize())

            // 3.2 - Merge sort between separators

            dim3 mergeSegmentGridSize(separators_per_sector, sector_qty);
            dim3 mergeSegmentBlockSize(t * 2, 1);

            merge_segments_kernel<<<mergeSegmentGridSize, mergeSegmentBlockSize>>>(separators_a_gpu, separators_b_gpu, src_gpu, dst_gpu, sector_size);
            CUDA_CHK(hipGetLastError())
            CUDA_CHK(hipDeviceSynchronize())

            int * aux = src_gpu;
            src_gpu = dst_gpu;
            dst_gpu = aux;
            swapped = !swapped;
        }

    }

    // 4 - Copy result to CPU

    CUDA_CHK( hipMemcpy(src_cpu, src_gpu, size, hipMemcpyDeviceToHost))
    CUDA_CHK ( hipFree(src_gpu) )
    CUDA_CHK ( hipFree(dst_gpu) )
    CUDA_CHK ( hipFree(separators_a_gpu) )
    CUDA_CHK ( hipFree(separators_b_gpu) )
}

void order_with_trust(int * src, int length){
    thrust::sort(src, src + length);
}
