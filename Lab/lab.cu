#include "hip/hip_runtime.h"
#include <stdio.h>

#include "include/lab.h"

#define WARP_SIZE 32
#define FULL_MASK 0xffffffff
#define NONE_MASK 0x00000000
#define RADIX_SORT_BLOCK_SIZE 256

__device__
int  exlusiveScan(int valor){
    int offset = 1;
    int lane = threadIdx.x;

    // 1º Etapa
    while (offset < WARP_SIZE ) {
        int preValor = __shfl_up_sync(FULL_MASK, valor, offset);
        if ( ((lane + 1) % (offset*2)) == 0 )
            valor += preValor;
        offset *= 2;
    }

    if (lane == WARP_SIZE - 1){
        valor = 0;
    }

    // 2º Etapa
    offset /= 2;
    while (offset > 0 ) {
        int valorOffsetSuperior = __shfl_down_sync(FULL_MASK, valor, offset);
        int valorOffsetInferior = __shfl_up_sync(FULL_MASK, valor, offset);

        bool moduloDobleOffset = ((lane + 1) % (offset*2)) == 0;
        valor += moduloDobleOffset ? valorOffsetInferior : 0;
        valor = !moduloDobleOffset && ((lane + 1) % (offset)) == 0 ? valorOffsetSuperior : valor;

        offset /= 2;
    }
    return valor;
}




/*
__device__
int split(int lane, int currentValue, int mask){



    //------------------

    int tid = threadIdx.x + threadIdx.y * blockDim.x; // tid in block
    int wid = tid % WARP_SIZE; // id in warp

    int value = array[wid]; // value to be ordered (stored in order to reuse the "array")
    prefix_array[wid] = !(bool)(array[wid] & mask); // true if the bit is 0, false if the bit is 1

    // prefix sum
    exlusiveScan(prefix_array, array, 0, 0);

    int total_false = array[WARP_SIZE - 1] + prefix_array[WARP_SIZE - 1]; // total number of false
    int order_index = prefix_array[wid] ? array[wid] : total_false + wid - array[wid]; // order index

    // write the value in the correct position
    array[order_index] = value;

    // check if the array is ordered
    value = array[wid];
    int nextValue = __shfl_down_sync(FULL_MASK, value, 1);

    if (wid < WARP_SIZE - 1 && value > nextValue) {
        ordered = false;
    }
}
*/

__global__
void radix_sort_kernel(int * src){

    __shared__ int swap [RADIX_SORT_BLOCK_SIZE];

    int lane = threadIdx.x % WARP_SIZE; //Se asume que los bloques son multiplos de 32, por lo que para obtener mi lane no presiso saber nada mas que mi modulo 32.
    int tie = threadIdx.x / WARP_SIZE;
    int idInArray = threadIdx.x + blockDim.x * blockIdx.x;
    int currentValue = src[idInArray];

    int valorThreadPrevio = __shfl_up_sync(FULL_MASK, currentValue, 1); //Obtengo el valor de mi lane previo
    if (lane == 0)
        valorThreadPrevio = currentValue - 1; //En caso que sea el primer lane, hago que mi "anterior" numero sea siempre menor, en particular uno menor

    int mask = 1;
    while (__all_sync(FULL_MASK,valorThreadPrevio <= currentValue) == 0){ //La operacion termina cuando el warp esta ordenado

        int notValidateMask = (currentValue & mask) == 0 ? 1 : 0;
        int valueInScan = exlusiveScan(notValidateMask);

        int totalFalse = notValidateMask + valueInScan; //Este valor es invalido para todos menos el ultimo lane, por eso en la siguiente linea todos se lo pide
        totalFalse = __shfl_sync(FULL_MASK, totalFalse, WARP_SIZE - 1); //Todos los lane obtiene el valor correcto del ultimo lane

        int t = lane - valueInScan + totalFalse;
        int nuevaPosicion = notValidateMask ? valueInScan : t;
        swap[tie + nuevaPosicion] = currentValue;

        __syncwarp();

        currentValue = swap[tie + lane];

        mask <<= 1; //Muevo la mascara

        valorThreadPrevio = __shfl_up_sync(FULL_MASK, currentValue, 1);
        if (lane == 0)
            valorThreadPrevio = currentValue - 1;
    }

    src[idInArray] = currentValue;
}






/**
 * Dado un array ordenado desde [posicionInicio, posicionInicio + size ], devuelve la posicion en la que deberia ser insertado objetoBuscado
 * Nota: si previoAIguales == true, este debuelbe la posicion de forma que objetoBuscado sea insertado antes que los iguales,
 *  en caso contrario sera la posicion de ser insertado detras de los iguales
 *
 *  Retorna la posicion relativa dentro del arreglo. O sea seria la posicion real dentro del array, menos el inicio
 * @param posicionInicio
 * @param size
 * @param objetoBuscado
 * @param previoAIguales
 * @return
 */

__device__
int busquedaPorBiparticion(int * array, int posicionInicio, int size, int objetoBuscado, bool previoAIguales){

    int final = posicionInicio + size - 1;
    int inicio = posicionInicio;

    int medio = (inicio + final) /2;

    while (inicio < final){ //Hay mas de un elemento en el area de busqueda

        int pivot = array[medio];

        bool buscarAbajo = previoAIguales ? objetoBuscado <= pivot : objetoBuscado < pivot;

        inicio = buscarAbajo ? inicio : medio + 1;
        final  = buscarAbajo ? medio  : final;
        medio  = (inicio + final) /2;

    }

    int pivot = array[medio];

    if (previoAIguales){
        return (pivot < objetoBuscado ? medio + 1 : medio) - posicionInicio;
    } else {
        return (pivot > objetoBuscado ? medio : medio + 1) - posicionInicio;
    }

}


/**
 * En base a dos array ordenados CONSECUTIVOS de largo largoA y largoB respectivamente
 * escribe en "array" el nuevo array ordenado producto de ordenar los dos anteriores
 * @param array
 * @param largoA
 * @param largoB
 */
__global__
void orderedJoin(int * src, int largoA, int largoB){
    extern __shared__ int shared[]; //Size = largoA + largob . Se almacena de forma compartida la informacion de los dos arrays a juntar
    //Voy a destinar los primeros "largoA" threads para ordenar los elementos de A, los demas para ordenar los de B
    bool soyDeB = threadIdx.x >= largoA;
    int idEnArray = soyDeB ? threadIdx.x - largoA: threadIdx.x; //Es la posicion dentro de A o B respectivamente
    int valor = src[ blockIdx.x * blockDim.x + threadIdx.x ]; //Todo analizar viabilidad de hacer consulta coalleced
    shared[threadIdx.x] = valor;

    __syncthreads();

    // Nota: Aqui hay que tener un cuidado adicional, ¿Que pasa si en la misma posiicon i de A y B tenemos que r(a_i, B) = r(b_i, A)?
    //         La solucion que encontramos es hacer que ese j no sea igual discriminando entre el array A y el B, dando la presetncia a A, mas detalle a continuacion:
    //         Notar que si a_i < b_i, entonces r(a_i, B) < r(b_i,B) = i = r(a_i, A) < r(b_i, A) (analogo para b_i < a_i), entonces aqui no hay problema.
    //         Pero lo mismo no ocurre si a_i = b_i = h, ya que dependiendo de la politica de la busqueda, tanto el a_i como el b_i tendrian como r(a_i, B) = r(b_i, A) = posicion al inicio (o final) de la rafaga de h
    //         Es por eso que se propone que la politica de busqueda de r(a_i, B) y r(b_i, A) sea diferente
    //         De forma tal que los elementos de A se "inserten" previo a todos los valores iguales en B, mientras que para los de B buscaremos su posicion de forma tal que sea luego de los iguales en A
    //         Esto ultimo es lo que representa el ultimo parametro de el metodo busquedaPorBiparticion

    int posicionEnElOtroArray = busquedaPorBiparticion(shared, soyDeB ? 0 : largoA, soyDeB ? largoA : largoB, valor, !soyDeB);

    __syncthreads();

    shared[idEnArray + posicionEnElOtroArray] = valor; //Escribo mi valor en la nueva posicion

    __syncthreads();

    src[ blockIdx.x * blockDim.x + threadIdx.x ] = shared[threadIdx.x];
}

void test_with_block_under_256(int * srcCpu, int length){
    int * srcGpu = NULL;

    size_t size = length * sizeof (int);
    CUDA_CHK( hipMalloc ((void **)& srcGpu , size ) )

    CUDA_CHK( hipMemcpy(srcGpu, srcCpu, size, hipMemcpyHostToDevice))

    //Etapa 1: Comienzo por el radixSort

    dim3 gridSizeRadixSort ( length / 32, 1);
    dim3 blockSizeRadixSort (32, 1);

    radix_sort_kernel<<<gridSizeRadixSort, blockSizeRadixSort>>>(srcGpu);

    CUDA_CHK(hipGetLastError())
    CUDA_CHK(hipDeviceSynchronize())

    //Etapa 2: Voy haciando merge sort hasta termino o llego a un bloque de 256

    int blockSize = 64; //32*2;
    while (blockSize <= min(256, length)){

        dim3 gridSizeOrdenerJoin ( length / blockSize, 1);
        dim3 blockSizeOrdererJoin (blockSize, 1);

        orderedJoin<<<gridSizeOrdenerJoin, blockSizeOrdererJoin, blockSize*sizeof(int)>>>(srcGpu, blockSize/2, blockSize/2);
        CUDA_CHK(hipGetLastError())
        CUDA_CHK(hipDeviceSynchronize())

        blockSize *=2;
    }

    CUDA_CHK( hipMemcpy(srcCpu, srcGpu, size, hipMemcpyDeviceToHost))
    CUDA_CHK ( hipFree(srcGpu) )

}


void test_radix_sort(int * srcCpu){
    int * srcGpu = NULL;

    //allocate
    size_t size = 64 * sizeof (int);
    CUDA_CHK( hipMalloc ((void **)& srcGpu , size ) )

    CUDA_CHK( hipMemcpy(srcGpu, srcCpu, size, hipMemcpyHostToDevice))


    dim3 gridSize ( 2, 1);
    dim3 blockSize (32, 1);

    radix_sort_kernel<<<gridSize, blockSize>>>(srcGpu);
    CUDA_CHK(hipGetLastError())
    CUDA_CHK(hipDeviceSynchronize())

    CUDA_CHK( hipMemcpy(srcCpu, srcGpu, size, hipMemcpyDeviceToHost))
    CUDA_CHK ( hipFree(srcGpu) )
}
