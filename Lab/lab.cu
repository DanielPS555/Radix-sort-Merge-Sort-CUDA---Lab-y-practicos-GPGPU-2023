#include "hip/hip_runtime.h"
#include <stdio.h>

#include "include/lab.h"

#define WARP_SIZE 32
#define FULL_MASK 0xffffffff
#define NONE_MASK 0x00000000
#define RADIX_SORT_BLOCK_SIZE 256

__device__
int  exlusiveScan(int valor){
    int offset = 1;
    int lane = threadIdx.x;

    // 1º Etapa
    while (offset < WARP_SIZE ) {
        int preValor = __shfl_up_sync(FULL_MASK, valor, offset);
        if ( ((lane + 1) % (offset*2)) == 0 )
            valor += preValor;
        offset *= 2;
    }

    if (lane == WARP_SIZE - 1){
        valor = 0;
    }

    // 2º Etapa
    offset /= 2;
    while (offset > 0 ) {
        int valorOffsetSuperior = __shfl_down_sync(FULL_MASK, valor, offset);
        int valorOffsetInferior = __shfl_up_sync(FULL_MASK, valor, offset);

        bool moduloDobleOffset = ((lane + 1) % (offset*2)) == 0;
        valor += moduloDobleOffset ? valorOffsetInferior : 0;
        valor = !moduloDobleOffset && ((lane + 1) % (offset)) == 0 ? valorOffsetSuperior : valor;

        offset /= 2;
    }
    return valor;
}




/*
__device__
int split(int lane, int currentValue, int mask){



    //------------------

    int tid = threadIdx.x + threadIdx.y * blockDim.x; // tid in block
    int wid = tid % WARP_SIZE; // id in warp

    int value = array[wid]; // value to be ordered (stored in order to reuse the "array")
    prefix_array[wid] = !(bool)(array[wid] & mask); // true if the bit is 0, false if the bit is 1

    // prefix sum
    exlusiveScan(prefix_array, array, 0, 0);

    int total_false = array[WARP_SIZE - 1] + prefix_array[WARP_SIZE - 1]; // total number of false
    int order_index = prefix_array[wid] ? array[wid] : total_false + wid - array[wid]; // order index

    // write the value in the correct position
    array[order_index] = value;

    // check if the array is ordered
    value = array[wid];
    int nextValue = __shfl_down_sync(FULL_MASK, value, 1);

    if (wid < WARP_SIZE - 1 && value > nextValue) {
        ordered = false;
    }
}
*/

__global__
void radix_sort_kernel(int * src){

    __shared__ int swap [RADIX_SORT_BLOCK_SIZE];

    int lane = threadIdx.x % WARP_SIZE; //Se asume que los bloques son multiplos de 32, por lo que para obtener mi lane no presiso saber nada mas que mi modulo 32.
    int tie = threadIdx.x / WARP_SIZE;
    int idInArray = threadIdx.x + blockDim.x * blockIdx.x;
    int currentValue = src[idInArray];

    int valorThreadPrevio = __shfl_up_sync(FULL_MASK, currentValue, 1); //Obtengo el valor de mi lane previo
    if (lane == 0)
        valorThreadPrevio = currentValue - 1; //En caso que sea el primer lane, hago que mi "anterior" numero sea siempre menor, en particular uno menor

    int mask = 1;
    while (__all_sync(FULL_MASK,valorThreadPrevio <= currentValue) == 0){ //La operacion termina cuando el warp esta ordenado

        int notValidateMask = (currentValue & mask) == 0 ? 1 : 0;
        int valueInScan = exlusiveScan(notValidateMask);

        int totalFalse = notValidateMask + valueInScan; //Este valor es invalido para todos menos el ultimo lane, por eso en la siguiente linea todos se lo pide
        totalFalse = __shfl_sync(FULL_MASK, totalFalse, WARP_SIZE - 1); //Todos los lane obtiene el valor correcto del ultimo lane

        int t = lane - valueInScan + totalFalse;
        int nuevaPosicion = notValidateMask ? valueInScan : t;
        swap[tie + nuevaPosicion] = currentValue;

        __syncwarp();

        currentValue = swap[tie + lane];

        mask <<= 1; //Muevo la mascara

        valorThreadPrevio = __shfl_up_sync(FULL_MASK, currentValue, 1);
        if (lane == 0)
            valorThreadPrevio = currentValue - 1;
    }

    src[idInArray] = currentValue;
}






/**
 * Dado un array ordenado desde [posicionInicio, posicionInicio + size ], devuelve la posicion en la que deberia ser insertado objetoBuscado
 * Nota: si previoAIguales == true, este devuelve la posicion de forma que objetoBuscado sea insertado antes que los iguales,
 *  en caso contrario sera la posicion de ser insertado detras de los iguales
 *
 *  Retorna la posicion relativa dentro del arreglo. O sea seria la posicion real dentro del array, menos el inicio
 * @param posicionInicio
 * @param size
 * @param objetoBuscado
 * @param previoAIguales
 * @return
 */

__device__
int busquedaPorBiparticion(int * array, int posicionInicio, int size, int objetoBuscado, bool previoAIguales){

    int final = posicionInicio + size - 1;
    int inicio = posicionInicio;

    int medio = (inicio + final) /2;

    while (inicio < final){ //Hay mas de un elemento en el area de busqueda

        int pivot = array[medio];

        bool buscarAbajo = previoAIguales ? objetoBuscado <= pivot : objetoBuscado < pivot;

        inicio = buscarAbajo ? inicio : medio + 1;
        final  = buscarAbajo ? medio  : final;
        medio  = (inicio + final) /2;

    }

    int pivot = array[medio];

    if (previoAIguales){
        return (pivot < objetoBuscado ? medio + 1 : medio) - posicionInicio;
    } else {
        return (pivot > objetoBuscado ? medio : medio + 1) - posicionInicio;
    }

}


/**
 * En base a dos array ordenados CONSECUTIVOS de largo largoA y largoB respectivamente
 * escribe en "array" el nuevo array ordenado producto de ordenar los dos anteriores
 * @param array
 * @param largoA
 * @param largoB
 */
__global__
void orderedJoin(int * src, int largoA, int largoB){
    extern __shared__ int shared[]; //Size = largoA + largob . Se almacena de forma compartida la informacion de los dos arrays a juntar
    //Voy a destinar los primeros "largoA" threads para ordenar los elementos de A, los demas para ordenar los de B
    bool soyDeB = threadIdx.x >= largoA;
    int idEnArray = soyDeB ? threadIdx.x - largoA: threadIdx.x; //Es la posicion dentro de A o B respectivamente
    int valor = src[ blockIdx.x * blockDim.x + threadIdx.x ]; //Todo analizar viabilidad de hacer consulta coalleced
    shared[threadIdx.x] = valor;

    __syncthreads();

    // Nota: Aqui hay que tener un cuidado adicional, ¿Que pasa si en la misma posiicon i de A y B tenemos que r(a_i, B) = r(b_i, A)?
    //         La solucion que encontramos es hacer que ese j no sea igual discriminando entre el array A y el B, dando la presetncia a A, mas detalle a continuacion:
    //         Notar que si a_i < b_i, entonces r(a_i, B) < r(b_i,B) = i = r(a_i, A) < r(b_i, A) (analogo para b_i < a_i), entonces aqui no hay problema.
    //         Pero lo mismo no ocurre si a_i = b_i = h, ya que dependiendo de la politica de la busqueda, tanto el a_i como el b_i tendrian como r(a_i, B) = r(b_i, A) = posicion al inicio (o final) de la rafaga de h
    //         Es por eso que se propone que la politica de busqueda de r(a_i, B) y r(b_i, A) sea diferente
    //         De forma tal que los elementos de A se "inserten" previo a todos los valores iguales en B, mientras que para los de B buscaremos su posicion de forma tal que sea luego de los iguales en A
    //         Esto ultimo es lo que representa el ultimo parametro de el metodo busquedaPorBiparticion

    int posicionEnElOtroArray = busquedaPorBiparticion(shared, soyDeB ? 0 : largoA, soyDeB ? largoA : largoB, valor, !soyDeB);

    __syncthreads();

    shared[idEnArray + posicionEnElOtroArray] = valor; //Escribo mi valor en la nueva posicion

    __syncthreads();

    src[ blockIdx.x * blockDim.x + threadIdx.x ] = shared[threadIdx.x];
}




/**
 * Threads will read
 */
 /*
__global__
void read_separators(int* a_in, int* b_in, int* s_out, int a_size, int b_size, int s_size, int t_size, int separator_count)
{
    // TODO: are s_size and separator_count the same?

    int section_id = blockIdx.x;  // A - B section
    int section_offset = section_id * separator_count;
    int separatorId = threadIdx.x + threadIdx.y * blockDim.x;

    if (separatorId >= separator_count)
        return;

    int a_index = separatorId * t_size;
    int b_index = a_index;

    if (separatorId < s_size)
    {
        // last segment gets the last element
        if ((s_size - 1) == separatorId)
        {
            a_index = a_size - 1;
            b_index = b_size - 1;
        }

        if (a_index < a_size)
            s_out[section_offset + separatorId] = a_in[a_index];

        if (index < b_size)
            s_out[section_offset + separatorId + separator_count] = b_in[b_index];
    }
} */


/**
 * Read separators from an input buffer.
 * @param data_in Input buffer
 * @param data_size Size of the input buffer
 * @param separators_out Output buffer with each separator
 * @param separators_size Size of the output buffer separators_out for all separators in all segments
 * @param sector_size Size of the sector A + B
 * @param separators_per_sector Total amount of separators per sector
 * The kernel will read both A and B buffers and will write the separators in the output buffer.
 */
__global__
void read_separators(int * data_in, size_t data_size, int * separators_out, size_t separators_size, int sector_size, int separators_per_sector) {
    // Asumamos por ahora que no hay casos donde el tamaño de los datos no sea multiplo del tamaño de la seccion
    //int separatorId = threadIdx.x + threadIdx.y * blockDim.x + SECTION_ID * SECTION_SIZE;
#define THREAD_ID threadIdx.x + threadIdx.y * blockDim.x
#define BLOCK_ID  blockIdx.x + blockIdx.y * gridDim.x
#define BLOCK_DIM blockDim.x * blockDim.y

    int segment_size = sector_size / separators_per_sector;

    // unique id for the thread
    int uid = THREAD_ID + BLOCK_ID * BLOCK_DIM;
    // based on the unique id we find the sector_id. The id of the AB sector
    int sector_id = uid / sector_size;
    // separator in the sector
    int separator_id = uid % separators_per_sector;
    // offset for the data array
    int sector_offset_d = sector_id * sector_size;

    // int a_index = separatorId * t_size + SECTION_ID * segment_size * 2;
    int a_index = sector_offset_d + separator_id * segment_size;
    // int b_index = a_index + segment_size;
    int b_index = a_index + sector_size / 2;

    // offset para acceder al array de separadores
    int sector_offset_s = separators_per_sector * sector_id;

    if (separator_id < separators_per_sector / 2 && sector_offset_s + separator_id + separators_per_sector / 2 < separators_size) {
        if (a_index < data_size)
            separators_out[sector_offset_s + separator_id] = data_in[a_index];

        if (b_index < data_size)
            separators_out[sector_offset_s + separator_id + separators_per_sector / 2] = data_in[b_index];
    }
}

void test_with_block_under_256(int * srcCpu, int length){
    int * srcGpu = NULL;

    size_t size = length * sizeof (int);
    CUDA_CHK( hipMalloc ((void **)& srcGpu , size ) )

    CUDA_CHK( hipMemcpy(srcGpu, srcCpu, size, hipMemcpyHostToDevice))

    //Etapa 1: Comienzo por el radixSort

    dim3 gridSizeRadixSort ( length / 32, 1);
    dim3 blockSizeRadixSort (32, 1);

    radix_sort_kernel<<<gridSizeRadixSort, blockSizeRadixSort>>>(srcGpu);

    CUDA_CHK(hipGetLastError())
    CUDA_CHK(hipDeviceSynchronize())

    //Etapa 2: Voy haciando merge sort hasta termino o llego a un bloque de 256

    int blockSize = 64; //32*2;
    while (blockSize <= min(256, length)){

        dim3 gridSizeOrdenerJoin ( length / blockSize, 1);
        dim3 blockSizeOrdererJoin (blockSize, 1);

        orderedJoin<<<gridSizeOrdenerJoin, blockSizeOrdererJoin, blockSize*sizeof(int)>>>(srcGpu, blockSize/2, blockSize/2);
        CUDA_CHK(hipGetLastError())
        CUDA_CHK(hipDeviceSynchronize())

        blockSize *=2;
    }

    CUDA_CHK( hipMemcpy(srcCpu, srcGpu, size, hipMemcpyDeviceToHost))
    CUDA_CHK ( hipFree(srcGpu) )
}

void test_secuence_reading (int * srcCpu, int length){

    int blockSize = 256; // se ejecuta test_with_block_under_256 antes

    int * srcGpu = NULL;

    size_t size = length * sizeof (int);
    CUDA_CHK( hipMalloc ((void **)& srcGpu , size ) )

    CUDA_CHK( hipMemcpy(srcGpu, srcCpu, size, hipMemcpyHostToDevice))

    if (blockSize <= length) {

        int segment_count = length / (256 / 2); // How many segments of 256/2 are there

        // so the seprators are always going to be the same, only their vales are going to change
        int* gpu_segment_values;
        CUDA_CHK( hipMalloc ((void **)& gpu_segment_values , segment_count * sizeof(int) ) )
        int* cpu_segment_values = (int*) malloc(segment_count * sizeof(int));

        // read separators
        // size of each A + B
        int sector_size = blockSize * 2;
        int t = blockSize / 2;


        //while (segment_size <= length) {


            // int section_qty = length / (blockSize * 2); // How many ab groups are there

            //dim3 gridSize(section_qty, 1);
            dim3 dimBlockSize(32, 32);
            dim3 gridSize((32 * 32 + segment_count - 1) / segment_count, 1);
            int separators_per_sector = sector_size / t;

            read_separators<<<gridSize, dimBlockSize>>>(srcGpu, length, gpu_segment_values, segment_count, sector_size, separators_per_sector);

            CUDA_CHK(hipGetLastError())
            CUDA_CHK(hipDeviceSynchronize())

            // foreach sector
            for (int i = 0; i < segment_count; i++) {
                printf("%d ", cpu_segment_values[i]);
            }


            CUDA_CHK( hipMemcpy(cpu_segment_values, gpu_segment_values, segment_count * sizeof(int), hipMemcpyDeviceToHost) )
            for (int i = 0; i < segment_count; i++) {
                printf("%d ", cpu_segment_values[i]);
            }



            // encuentro separadores
            // Sa + Sb
            // sector_size *= 2;

        //}



        printf("\n");

        free(cpu_segment_values);
        CUDA_CHK( hipFree(gpu_segment_values) )

    }

    CUDA_CHK( hipMemcpy(srcCpu, srcGpu, size, hipMemcpyDeviceToHost))
    CUDA_CHK ( hipFree(srcGpu) )
}




void test_radix_sort(int * srcCpu){
    int * srcGpu = NULL;

    //allocate
    size_t size = 64 * sizeof (int);
    CUDA_CHK( hipMalloc ((void **)& srcGpu , size ) )

    CUDA_CHK( hipMemcpy(srcGpu, srcCpu, size, hipMemcpyHostToDevice))


    dim3 gridSize ( 2, 1);
    dim3 blockSize (32, 1);

    radix_sort_kernel<<<gridSize, blockSize>>>(srcGpu);
    CUDA_CHK(hipGetLastError())
    CUDA_CHK(hipDeviceSynchronize())

    CUDA_CHK( hipMemcpy(srcCpu, srcGpu, size, hipMemcpyDeviceToHost))
    CUDA_CHK ( hipFree(srcGpu) )
}
