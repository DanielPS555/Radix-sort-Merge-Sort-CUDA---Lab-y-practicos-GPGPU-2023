#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "math.h"
#include <algorithm>

#include "include/histogram.h"

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define MS(f,elap)                                                                                           \
        double elap=0;                                                                                       \
        {                                                                                                    \
        struct timespec t_ini,t_fin;                                                                         \
            clock_gettime(CLOCK_MONOTONIC, &t_ini);                                                          \
            f;                                                                                               \
            clock_gettime(CLOCK_MONOTONIC, &t_fin);                                                          \
            elap = 1000 * (t_fin.tv_sec - t_ini.tv_sec) + (t_fin.tv_nsec - t_ini.tv_nsec)/1000000.0;         \
        }

// Kernels
__global__ void simple_histogram_kernel(float *img_gpu_in, float *img_gpu_out, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float color = COLOR_SIZE; // Out of range
    if (x < width && y < height) {
        color = img_gpu_in[x + y * width];
    }

    if (color < (float)COLOR_SIZE) {
        atomicAdd(&img_gpu_out[(int)color], 1.f);
    }
}

__global__ void shared_memory_histogram_kernel(float *img_gpu_in, float *img_gpu_out, int width, int height) {
    __shared__ float h_block[COLOR_SIZE];

    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (tid < COLOR_SIZE) {
        h_block[tid] = 0.f;
    }

    __syncthreads();

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float color = COLOR_SIZE; // Out of range
    if (x < width && y < height) {
        color = img_gpu_in[x + y * width];
    }

    if (color < (float)COLOR_SIZE) {
        atomicAdd(&h_block[(int)color], 1.f);
    }

    __syncthreads();

    if (tid < COLOR_SIZE) {
        atomicAdd(&img_gpu_out[tid], h_block[tid]);
    }
}


// Kernel callers
void gpu_execute_kernel(algorithm_type algo, const dim3 &gridSize, const dim3 &blockSize, float *img_gpu_in, float *img_gpu_out, int width, int height) {
    switch (algo) {
        case SIMPLE_HISTOGRAM:
            simple_histogram_kernel<<<gridSize, blockSize>>>(img_gpu_in, img_gpu_out, width, height);
            break;
        case SHARED_MEMORY_HISTOGRAM:
            shared_memory_histogram_kernel<<<gridSize, blockSize>>>(img_gpu_in, img_gpu_out, width, height);
            break;
        case IMPROVED_SHARED_MEMORY_HISTOGRAM:
            // improved_transpose_dummy_kernel<<<gridSize, blockSize>>>(img_gpu_in, img_gpu_out, width, height);
            break;
    }
    CUDA_CHK(hipGetLastError())
    CUDA_CHK(hipDeviceSynchronize())
}

// Utils
void allocate_and_copy_gpu(float* &gpu_in, float* &gpu_out, float *cpu_in, float *cpu_out, int width, int height) {
    size_t size = width * height * sizeof(float);
    size_t hist_size = COLOR_SIZE * sizeof(float);

    CUDA_CHK ( hipMalloc((void**)& gpu_in, size) )

    // Initialize gpu_out in 0
    CUDA_CHK ( hipMalloc((void**)& gpu_out, hist_size) )

    CUDA_CHK ( hipMemcpy(gpu_in, cpu_in, size, hipMemcpyHostToDevice) )
    CUDA_CHK ( hipMemcpy(gpu_out, cpu_out, hist_size, hipMemcpyHostToDevice) )

    CUDA_CHK ( hipMemset(gpu_out, 0, hist_size) )
}

void copy_and_free_gpu(float* &gpu_in, float* &gpu_out, float *cpu_out, int width, int height) {
    size_t hist_size = COLOR_SIZE * sizeof(float);
    CUDA_CHK ( hipMemcpy(cpu_out, gpu_out, hist_size, hipMemcpyDeviceToHost) )
    CUDA_CHK ( hipFree(gpu_in) )
    CUDA_CHK ( hipFree(gpu_out) )
}

double execute_kernel(algorithm_type algo, float* in_cpu_m, float* out_cpu_m, int width, int height) {
    float * img_gpu = NULL, * img_gpu_out = NULL;
    allocate_and_copy_gpu(img_gpu, img_gpu_out, in_cpu_m, out_cpu_m, width, height);

    // TODO: Assume the image is multiple of BLOCK_SIZE
    dim3 gridSize( (int)((float)width)/BLOCK_SIZE, (int)((float)height)/BLOCK_SIZE ); // 40 x 30
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

    MS(gpu_execute_kernel(algo, gridSize, blockSize, img_gpu, img_gpu_out, width, height), time)

    copy_and_free_gpu(img_gpu, img_gpu_out, out_cpu_m, width, height);

    return time;
}
