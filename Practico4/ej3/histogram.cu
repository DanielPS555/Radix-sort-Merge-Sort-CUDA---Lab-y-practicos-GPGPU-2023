#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "math.h"
#include <algorithm>

#include "include/histogram.h"

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define MS(f,elap)                                                                                           \
        double elap=0;                                                                                       \
        {                                                                                                    \
        struct timespec t_ini,t_fin;                                                                         \
            clock_gettime(CLOCK_MONOTONIC, &t_ini);                                                          \
            f;                                                                                               \
            clock_gettime(CLOCK_MONOTONIC, &t_fin);                                                          \
            elap = 1000 * (t_fin.tv_sec - t_ini.tv_sec) + (t_fin.tv_nsec - t_ini.tv_nsec)/1000000.0;         \
        }

// Kernels
__global__ void simple_histogram_kernel(float *img_gpu_in, float *img_gpu_out, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int color = (int)img_gpu_in[x + y * width];
        atomicAdd(&img_gpu_out[color], 1.f);
    }
}

__global__ void shared_memory_histogram_kernel(float *img_gpu_in, float *img_gpu_out, int width, int height) {
    __shared__ float h_block[COLOR_SIZE];

    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (tid < COLOR_SIZE) {
        h_block[tid] = 0.f;
    }

    __syncthreads();

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int color = (int)img_gpu_in[x + y * width];
        atomicAdd(&h_block[color], 1.f);
    }

    __syncthreads();

    if (tid < COLOR_SIZE) {
        atomicAdd(&img_gpu_out[tid], h_block[tid]);
    }
}

__global__ void matrix_histogram_kernel(float *img_gpu_in, float *img_gpu_out, int width, int height) {
    __shared__ float h_block[COLOR_SIZE];

    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;
    if (tid < COLOR_SIZE) {
        h_block[tid] = 0.f;
    }

    __syncthreads();

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int color = (int)img_gpu_in[x + y * width];
        atomicAdd(&h_block[color], 1.f);
    }

    __syncthreads();

    if (tid < COLOR_SIZE) {
        img_gpu_out[tid + bid * COLOR_SIZE] = h_block[tid];
    }
}

#define COLOR_PER_BLOCK 4
#define REDUCE_BLOCK_HEIGHT 128

__global__ void matrix_histogram_reduce_kernel(float *i_histogram_m, float *o_histogram_m, size_t height, size_t width, size_t size) {
    extern __shared__ float h_block[];

    // Initialize shared memory
    size_t tid_init = threadIdx.x + threadIdx.y * blockDim.x;
    if (tid_init < size) {
        h_block[tid_init] = 0.f;
        h_block[tid_init + blockDim.x * blockDim.y] = 0.f;
    }

    __syncthreads();

    // Copy to shared memory
    // width should be 256 color size
    size_t g_mem_a = blockIdx.y * blockDim.y * width * 2 + threadIdx.y * width + threadIdx.x + blockIdx.x * blockDim.x;
    size_t g_mem_b = g_mem_a + width * blockDim.y;

    size_t tid = threadIdx.x * blockDim.y * 2 + threadIdx.y; // tid on the shared block (transposed)

    if (g_mem_a < height * width) {
        h_block[tid] = i_histogram_m[g_mem_a];
    }
    if (g_mem_b < height * width) {
        h_block[tid + blockDim.y] = i_histogram_m[g_mem_b];
    }

    __syncthreads();

    // Reduce
    size_t reduce_width = fminf(blockDim.y * 2, height);
    int j = reduce_width;
    while (j > 1)
    {
        reduce_width = (j + 1) / 2; // integer ceil
        if (threadIdx.y < reduce_width && (threadIdx.y + reduce_width) < j) {
            h_block[tid] += h_block[tid + reduce_width];
        }
        j = reduce_width;
        __syncthreads();
    }

    __syncthreads();

    int color = threadIdx.x + blockIdx.x * blockDim.x;
    // Write to global memory
    if (threadIdx.y == 0) {
        o_histogram_m[blockIdx.y * 256 + color] = h_block[tid];
    }

}

// Kernel callers
void gpu_execute_kernel(algorithm_type algo, const dim3 &gridSize, const dim3 &blockSize, float *img_gpu_in, float *img_gpu_out, int width, int height) {
    switch (algo) {
        case SIMPLE_HISTOGRAM:
            simple_histogram_kernel<<<gridSize, blockSize>>>(img_gpu_in, img_gpu_out, width, height);
            break;
        case SHARED_MEMORY_HISTOGRAM:
            shared_memory_histogram_kernel<<<gridSize, blockSize>>>(img_gpu_in, img_gpu_out, width, height);
            break;
        default:
            break;
    }
    CUDA_CHK(hipGetLastError())
    CUDA_CHK(hipDeviceSynchronize())
}

// Utils
void allocate_and_copy_gpu(float* &gpu_in, float* &gpu_out, float *cpu_in, float *cpu_out, int width, int height) {
    size_t size = width * height * sizeof(float);
    size_t hist_size = COLOR_SIZE * sizeof(float);

    CUDA_CHK ( hipMalloc((void**)& gpu_in, size) )

    // Initialize gpu_out in 0
    CUDA_CHK ( hipMalloc((void**)& gpu_out, hist_size) )

    CUDA_CHK ( hipMemcpy(gpu_in, cpu_in, size, hipMemcpyHostToDevice) )
    CUDA_CHK ( hipMemcpy(gpu_out, cpu_out, hist_size, hipMemcpyHostToDevice) )

    CUDA_CHK ( hipMemset(gpu_out, 0, hist_size) )
}

void copy_and_free_gpu(float* &gpu_in, float* &gpu_out, float *cpu_out, int width, int height) {
    size_t hist_size = COLOR_SIZE * sizeof(float);
    CUDA_CHK ( hipMemcpy(cpu_out, gpu_out, hist_size, hipMemcpyDeviceToHost) )
    CUDA_CHK ( hipFree(gpu_in) )
    CUDA_CHK ( hipFree(gpu_out) )
}

double execute_kernel(algorithm_type algo, float* in_cpu_m, float* out_cpu_m, int width, int height) {
    float * img_gpu = NULL, * img_gpu_out = NULL;
    allocate_and_copy_gpu(img_gpu, img_gpu_out, in_cpu_m, out_cpu_m, width, height);

    // TODO: Assume the image is multiple of BLOCK_SIZE
    dim3 gridSize( (int)((float)width)/BLOCK_SIZE, (int)((float)height)/BLOCK_SIZE ); // 40 x 30
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

    MS(gpu_execute_kernel(algo, gridSize, blockSize, img_gpu, img_gpu_out, width, height), time)

    copy_and_free_gpu(img_gpu, img_gpu_out, out_cpu_m, width, height);

    return time;
}

/**
 * This is for the kernel with a matrix of histogram (ex b)
 */

#define COLORS_PER_BLOCK 4
#define BLOCKS_PER_BLOCK 64  // Must be divisible by 2

bool execute_kernel_histogram(int width, int height, float * img_gpu, float * img_gpu_hist, float * img_gpu_hist_b) {
    // Grid
    dim3 gridSize( (int)((float)width)/BLOCK_SIZE, (int)((float)height)/BLOCK_SIZE );
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    size_t block_qty = (int)((float)width)/BLOCK_SIZE * (int)((float)height)/BLOCK_SIZE;
    // Execute
    // Create histogram matrix
    matrix_histogram_kernel<<<gridSize, blockSize>>>(img_gpu, img_gpu_hist, width, height);

    bool swapped = false;
    int reduceWidth = COLOR_SIZE;
    int reduceHeight = block_qty;

    dim3 reduceBlockSize(COLORS_PER_BLOCK, BLOCKS_PER_BLOCK / 2);
    size_t sharedMemSize = BLOCKS_PER_BLOCK * COLORS_PER_BLOCK * sizeof(int);

    do
    {
        dim3 reduceGridSize( (int)ceil(((float)reduceWidth)/COLORS_PER_BLOCK), (int)ceil(((float)reduceHeight)/BLOCKS_PER_BLOCK)); // Multiplied 2 because each block sums "another" block

        CUDA_CHK(hipDeviceSynchronize())
        if (!swapped)
            matrix_histogram_reduce_kernel<<<reduceGridSize, reduceBlockSize, sharedMemSize>>>(img_gpu_hist, img_gpu_hist_b, reduceHeight, reduceWidth, block_qty * 256);
        else
            matrix_histogram_reduce_kernel<<<reduceGridSize, reduceBlockSize, sharedMemSize>>>(img_gpu_hist_b, img_gpu_hist, reduceHeight, reduceWidth, block_qty * 256);

        CUDA_CHK(hipGetLastError())

        swapped = !swapped;

        reduceHeight = ceil((float)reduceHeight / BLOCKS_PER_BLOCK);

    } while (1 < reduceHeight);

    CUDA_CHK(hipDeviceSynchronize())

    return swapped;
}

double execute_histogram(float* in_cpu_m, float* out_cpu_m, int width, int height) {
    // img_gpu is the same as in previous exercises, img_gpu_out should be a matrix of histograms 256 * block_qty
    float * img_gpu = NULL, * img_gpu_hist = NULL, *img_gpu_hist_b = NULL;

    size_t in_size = width * height * sizeof(float);
    size_t block_qty = (int)((float)width)/BLOCK_SIZE * (int)((float)height)/BLOCK_SIZE;
    size_t hist_size = block_qty * COLOR_SIZE * sizeof(float);

    // Allocate
    CUDA_CHK ( hipMalloc((void**)& img_gpu, in_size) )
    CUDA_CHK ( hipMalloc((void**)& img_gpu_hist, hist_size) )
    CUDA_CHK ( hipMalloc((void**)& img_gpu_hist_b, hist_size) )
    CUDA_CHK ( hipMemcpy(img_gpu, in_cpu_m, in_size, hipMemcpyHostToDevice) )
    CUDA_CHK ( hipMemset(img_gpu_hist, 0, hist_size) )  // Initialize gpu_out in 0

    bool swapped;
    MS(swapped = execute_kernel_histogram(width, height, img_gpu, img_gpu_hist, img_gpu_hist_b), time)

    if (!swapped)
        CUDA_CHK ( hipMemcpy(out_cpu_m, img_gpu_hist, 256 * sizeof(int), hipMemcpyDeviceToHost) )
    else
        CUDA_CHK ( hipMemcpy(out_cpu_m, img_gpu_hist_b, 256 * sizeof(int), hipMemcpyDeviceToHost) )

    CUDA_CHK ( hipFree(img_gpu) )
    CUDA_CHK ( hipFree(img_gpu_hist) )
    CUDA_CHK ( hipFree(img_gpu_hist_b) )

    return time;
}