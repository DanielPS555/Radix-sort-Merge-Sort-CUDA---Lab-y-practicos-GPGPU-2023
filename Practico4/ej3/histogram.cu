#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "math.h"
#include <algorithm>

#include "include/histogram.h"

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define MS(f,elap)                                                                                           \
        double elap=0;                                                                                       \
        {                                                                                                    \
        struct timespec t_ini,t_fin;                                                                         \
            clock_gettime(CLOCK_MONOTONIC, &t_ini);                                                          \
            f;                                                                                               \
            clock_gettime(CLOCK_MONOTONIC, &t_fin);                                                          \
            elap = 1000 * (t_fin.tv_sec - t_ini.tv_sec) + (t_fin.tv_nsec - t_ini.tv_nsec)/1000000.0;         \
        }

// Kernels
__global__ void simple_histogram_kernel(float *img_gpu_in, float *img_gpu_out, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int color = (int)img_gpu_in[x + y * width];
        atomicAdd(&img_gpu_out[color], 1.f);
    }
}

__global__ void shared_memory_histogram_kernel(float *img_gpu_in, float *img_gpu_out, int width, int height) {
    __shared__ float h_block[COLOR_SIZE];

    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (tid < COLOR_SIZE) {
        h_block[tid] = 0.f;
    }

    __syncthreads();

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int color = (int)img_gpu_in[x + y * width];
        atomicAdd(&h_block[color], 1.f);
    }

    __syncthreads();

    if (tid < COLOR_SIZE) {
        atomicAdd(&img_gpu_out[tid], h_block[tid]);
    }
}

__global__ void matrix_histogram_kernel(float *img_gpu_in, float *img_gpu_out, int width, int height) {
    __shared__ float h_block[COLOR_SIZE];

    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;
    if (tid < COLOR_SIZE) {
        h_block[tid] = 0.f;
    }

    __syncthreads();

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int color = (int)img_gpu_in[x + y * width];
        atomicAdd(&h_block[color], 1.f);
    }

    __syncthreads();

    if (tid < COLOR_SIZE) {
        img_gpu_out[tid + bid * COLOR_SIZE] = h_block[tid];
    }
}

#define COLOR_PER_BLOCK 4
#define REDUCE_BLOCK_HEIGHT 128

__global__ void matrix_histogram_reduce_kernel(float *i_histogram_m, float *o_histogram_m, size_t height, size_t width) {
    extern __shared__ float h_block[]; // Color size * reduce size

    size_t reduce_height = blockDim.x;
    size_t reduce_width = blockDim.y * 2;

    size_t color_per_block = blockDim.x;
    size_t tid = threadIdx.x * reduce_width + threadIdx.y; // tid on the shared block (transposed)
    size_t color_id = threadIdx.x % color_per_block;

    // Copy to shared memory
    size_t g_mem_a = blockIdx.y * width * blockDim.x * 2 + threadIdx.y * width + threadIdx.x;
    size_t g_mem_b = g_mem_a + width * blockDim.y;

    h_block[tid] = i_histogram_m[g_mem_a];
    h_block[tid + blockDim.y] = i_histogram_m[g_mem_b];

    __syncthreads();

    // Reduce
    int i = reduce_width / 2;
    while (i > 0) {
        if (threadIdx.y < i) {
            h_block[tid] += h_block[tid + i];
        }
        __syncthreads();
        i /= 2;
    }

    // Write to global memory
    if (threadIdx.y == 0) {
        o_histogram_m[blockIdx.y * color_per_block + color_id] = h_block[tid];
    }

}

// Kernel callers
void gpu_execute_kernel(algorithm_type algo, const dim3 &gridSize, const dim3 &blockSize, float *img_gpu_in, float *img_gpu_out, int width, int height) {
    switch (algo) {
        case SIMPLE_HISTOGRAM:
            simple_histogram_kernel<<<gridSize, blockSize>>>(img_gpu_in, img_gpu_out, width, height);
            break;
        case SHARED_MEMORY_HISTOGRAM:
            shared_memory_histogram_kernel<<<gridSize, blockSize>>>(img_gpu_in, img_gpu_out, width, height);
            break;
        case IMPROVED_SHARED_MEMORY_HISTOGRAM:
            matrix_histogram_kernel<<<gridSize, blockSize>>>(img_gpu_in, img_gpu_out, width, height);
            break;
    }
    CUDA_CHK(hipGetLastError())
    CUDA_CHK(hipDeviceSynchronize())
}

// Utils
void allocate_and_copy_gpu(float* &gpu_in, float* &gpu_out, float *cpu_in, float *cpu_out, int width, int height) {
    size_t size = width * height * sizeof(float);
    size_t hist_size = COLOR_SIZE * sizeof(float);

    CUDA_CHK ( hipMalloc((void**)& gpu_in, size) )

    // Initialize gpu_out in 0
    CUDA_CHK ( hipMalloc((void**)& gpu_out, hist_size) )

    CUDA_CHK ( hipMemcpy(gpu_in, cpu_in, size, hipMemcpyHostToDevice) )
    CUDA_CHK ( hipMemcpy(gpu_out, cpu_out, hist_size, hipMemcpyHostToDevice) )

    CUDA_CHK ( hipMemset(gpu_out, 0, hist_size) )
}

void copy_and_free_gpu(float* &gpu_in, float* &gpu_out, float *cpu_out, int width, int height) {
    size_t hist_size = COLOR_SIZE * sizeof(float);
    CUDA_CHK ( hipMemcpy(cpu_out, gpu_out, hist_size, hipMemcpyDeviceToHost) )
    CUDA_CHK ( hipFree(gpu_in) )
    CUDA_CHK ( hipFree(gpu_out) )
}

double execute_kernel(algorithm_type algo, float* in_cpu_m, float* out_cpu_m, int width, int height) {
    float * img_gpu = NULL, * img_gpu_out = NULL;
    allocate_and_copy_gpu(img_gpu, img_gpu_out, in_cpu_m, out_cpu_m, width, height);

    // TODO: Assume the image is multiple of BLOCK_SIZE
    dim3 gridSize( (int)((float)width)/BLOCK_SIZE, (int)((float)height)/BLOCK_SIZE ); // 40 x 30
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

    MS(gpu_execute_kernel(algo, gridSize, blockSize, img_gpu, img_gpu_out, width, height), time)

    copy_and_free_gpu(img_gpu, img_gpu_out, out_cpu_m, width, height);

    return time;
}

/**
 * This is for the kernel with a matrix of histogram (ex b)
 */


double execute_kernel_histogram(float* in_cpu_m, float* out_cpu_m, int width, int height) {
    // img_gpu is the same as in previous exercises, img_gpu_out should be a matrix of histograms 256 * block_qty
    float * img_gpu = NULL, * img_gpu_hist = NULL;

    size_t in_size = width * height * sizeof(float);
    size_t block_qty = (int)((float)width)/BLOCK_SIZE * (int)((float)height)/BLOCK_SIZE;
    size_t hist_size = block_qty * COLOR_SIZE * sizeof(float);

    // Allocate
    CUDA_CHK ( hipMalloc((void**)& img_gpu, in_size) )
    CUDA_CHK ( hipMalloc((void**)& img_gpu_hist, hist_size) )
    CUDA_CHK ( hipMemcpy(img_gpu, in_cpu_m, in_size, hipMemcpyHostToDevice) )
    CUDA_CHK ( hipMemset(img_gpu_hist, 0, hist_size) )  // Initialize gpu_out in 0

    // Grid
    dim3 gridSize( (int)((float)width)/BLOCK_SIZE, (int)((float)height)/BLOCK_SIZE );
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

    // Execute



}